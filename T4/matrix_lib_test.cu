/*Lívia Lutz dos Santos - 2211055
Ana Luiza Pinto Marques - 2211960*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <hip/hip_runtime.h>
extern "C" {
#include "timer.h"
}
#include "matrix_lib.h"

float scalar_value = 0.0f;

struct matrix matrixA, matrixB, matrixC;


int store_matrix(struct matrix *matrix, char *filename) {
    FILE* arq = fopen(filename, "wb");

    if(arq == NULL){
        printf("Erro ao abrir o arquivo para escrita\n");
        return 0;
    }
    int qtd = fwrite(matrix->h_rows, sizeof(float), matrix->height*matrix->width, arq);

    if(qtd != matrix->height*matrix->width){
        printf("Erro ao escrever no arquivo\n");
        return 0;
    }

    fclose(arq);

    return 1;
}

int load_matrix(struct matrix *matrix, char *filename) {
    FILE* arq = fopen(filename, "rb");

    if(arq == NULL){
        printf("Erro ao abrir o arquivo\n");
        return 0;
    }

    int qtd = fread(matrix->h_rows, sizeof(float), matrix->height*matrix->width, arq);

    if(qtd != matrix->height*matrix->width){
        printf("Erro ao ler o arquivo\n");
        return 0;
    }

    fclose(arq);

    return 1;
}

int initialize_matrix(struct matrix *matrix, float value, float inc) {
    int ind;
    for(int i = 0; i < matrix->height; i++){
        for(int j = 0; j < matrix->width; j++){
            ind = i * matrix->height + j;
            matrix->h_rows[ind] = value;
            value += inc;
        }
    }

    return 1;
}

int print_matrix(struct matrix *matrix) {
    for(int i = 0; i < matrix->height; i++){
        for(int j = 0; j < matrix->width; j++){
	        if((i * matrix->height + j) > 256){
		        printf("Ooops...256 printing limit found...skipping printing...\n");
		        return 1;
	        }	
            printf("%f ", matrix->h_rows[i * matrix->height + j]);
        }
        printf("\n");
    }

    return 1;
}

int check_errors(struct matrix *matrix, float scalar_value) {
    for(int i = 0; i < matrix->height; i++){
        for(int j = 0; j < matrix->width; j++){
            if(matrix->h_rows[i * matrix->height + j] != scalar_value){
                printf("Matrix error\nExpected value : %f\nReceived value:%f\n", scalar_value, matrix->h_rows[i * matrix->height + j]);
                return 0;
            }
        }
    }

    return 1;
}

int main(int argc, char *argv[]) {
    char *result1_filename, *result2_filename;
    char *eptr = NULL;
    struct timeval start, stop, overall_t1, overall_t2;
    int carregaA, carregaB, inicializaC,max_mem_gpu,threads_per_block,max_blocks_per_grid,somaTotalMemMatriz,somaTotalMemB;
    hipError_t hipError_t;

    // Mark overall start time
    gettimeofday(&overall_t1, NULL);

    // Disable buffering entirely
    setbuf(stdout, NULL);

    // Check arguments
    if (argc != 13) {
            printf("Usage: %s <scalar_value> <DimA_M> <DimA_N> <DimB_M> <DimB_N> <NumThreadsBloco> <MaxBlocoGrid> <QtdMaxMem> <matrixA_filename> <matrixB_filename> <result1_filename> <result2_filename>\n", argv[0]);
            return 0;
    }

    // Convert arguments
    scalar_value = strtof(argv[1], &eptr);
    matrixA.height = strtol(argv[2], &eptr, 10);
    matrixA.width = strtol(argv[3], &eptr, 10);
    matrixB.height = strtol(argv[4], &eptr, 10);
    matrixB.width = strtol(argv[5], &eptr, 10);
    threads_per_block = strtol(argv[6], &eptr, 10);
    max_blocks_per_grid = strtol(argv[7], &eptr, 10);
    max_mem_gpu = strtol(argv[8], &eptr, 10);
    max_mem_gpu = max_mem_gpu * 1024 * 1024;

    matrixC.height = matrixA.height;
    matrixC.width = matrixB.width;

    result1_filename = argv[11];
    result2_filename = argv[12];

    int limite = set_grid_size(threads_per_block,max_blocks_per_grid);

    if(limite == 0){
        printf("Erro ao setar o tamanho do grid\n");
        return 0;
    }
  
    /* Allocate the arrays of the four matrixes */

    matrixA.h_rows = (float*) malloc((matrixA.height * matrixA.width) * sizeof(float));
    matrixB.h_rows = (float*) malloc((matrixB.height * matrixB.width) * sizeof(float));
    matrixC.h_rows = (float*) malloc((matrixA.height * matrixB.width) * sizeof(float));

    /*Checks allocations*/
    if(matrixA.h_rows == NULL || matrixB.h_rows == NULL || matrixC.h_rows == NULL){
        printf("Erro ao alocar memoria\n");
        return 0;
    }

    //Alocando as matrizes na GPU
    hipError_t = hipMalloc(&matrixA.d_rows, (matrixA.height * matrixA.width) * sizeof(float));
    if (hipError_t != hipSuccess) {
        printf("hipMalloc matrixA.d_rows returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
        return 1;
    }

    hipError_t = hipMalloc(&matrixB.d_rows, (matrixB.height * matrixB.width) * sizeof(float));
    if (hipError_t != hipSuccess) {
        printf("hipMalloc matrixB.d_rows returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
        return 1;
    }

    hipError_t = hipMalloc(&matrixC.d_rows, (matrixC.height * matrixC.width) * sizeof(float));
    if (hipError_t != hipSuccess) {
        printf("hipMalloc matrixC.d_rows returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
        return 1;
    }

    /* Initialize the three matrixes */
    carregaA = load_matrix(&matrixA, argv[9]);
    carregaB = load_matrix(&matrixB, argv[10]);

    /*Checks if matrixes were loaded correctly */
    if(carregaA == 0 || carregaB == 0){
        printf("Erro ao carregar as matrizes\n");
        return 0;
    }

    inicializaC = initialize_matrix(&matrixC, 0.0f, 0.0f);

    if(inicializaC != 1){
        printf("Erro ao inicializar a matriz C\n");
        return 0;
    }

    somaTotalMemMatriz = matrixA.height * matrixA.width * sizeof(float) + matrixB.height * matrixB.width * sizeof(float) + matrixC.height * matrixC.width * sizeof(float);
    somaTotalMemB = matrixB.height * matrixB.width * sizeof(float);

    /*Se for viável fazer a alocação completa da três matrizes na CPU e na GPGPU, o programa deve
    atribuir o valor FULL_ALLOCATION no campo alloc_mode da três matrizes.
    */
    if(max_mem_gpu >= somaTotalMemMatriz){
        matrixA.alloc_mode = 1;
        matrixB.alloc_mode = 1;
        matrixC.alloc_mode = 1;
    }

    /* Se não for viável fazer a alocação simultânea e completa das matrizes A, B e C na GPGPU, o
    programa deve tentar alocar simultaneamente a matriz B por completo e o equivalente a uma das
    linhas da matriz A e uma das linhas da matriz C na GPGPU. Se tiver sucesso nessa alocação, o
    programa deve atribuir o valor FULL_ALLOCATION no campo alloc_mode da matriz B e o valor
    PARTIAL_ALLOC no campo alloc_mode das matrizes A e C.
    */
    else if(max_mem_gpu < somaTotalMemMatriz && max_mem_gpu >= somaTotalMemB){
        
        //Alocando a matriz B na GPU por completo
        hipError_t = hipMemcpy(matrixB.d_rows, matrixB.h_rows, (matrixB.height * matrixB.width) * sizeof(float), hipMemcpyHostToDevice);
        if (hipError_t != hipSuccess) {
            printf("hipMemcpy matrixB.h_rows -> matrixB.d_rows returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
            return 1;
        }

        //alocando 1 linha de A na GPU
        for(int i = 0; i < matrixA.width; i++){
            hipError_t = hipMemcpy(matrixA.d_rows + i, matrixA.h_rows + i, sizeof(float), hipMemcpyHostToDevice);
            if (hipError_t != hipSuccess) {
                printf("hipMemcpy matrixA.h_rows -> matrixA.d_rows returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
                return 1;
            }
        }

        //alocando 1 linha de C na GPU
        for(int j = 0;j < matrixC.width;j++){
            hipError_t = hipMemcpy(matrixC.d_rows + j, matrixC.h_rows + j, sizeof(float), hipMemcpyHostToDevice);
            if (hipError_t != hipSuccess) {
                printf("hipMemcpy matrixC.h_rows -> matrixC.d_rows returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
                return 1;
            }
        }

        matrixA.alloc_mode = 0;
        matrixB.alloc_mode = 1;
        matrixC.alloc_mode = 0;
    }

    /*Se não for viável fazer a alocação completa da matriz B e a alocação parcial das matrizes A e C
    simultaneamente na GPGPU, o programa principal deve emitir uma notificação de erro de alocação
    de memória na GPGPU e encerrar sua execução.*/
    else{
        printf("Erro de alocação de memória na GPGPU\n");
        return 0;
    }

    //Alocando as matrizes na GPU por completo

    if(matrixA.alloc_mode == 1 && matrixB.alloc_mode == 1 && matrixC.alloc_mode == 1){
        hipError_t = hipMemcpy(matrixA.d_rows, matrixA.h_rows, (matrixA.height * matrixA.width) * sizeof(float), hipMemcpyHostToDevice);
        if (hipError_t != hipSuccess) {
            printf("hipMemcpy matrixA.h_rows -> matrixA.d_rows returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
            return 1;
        }

        hipError_t = hipMemcpy(matrixB.d_rows, matrixB.h_rows, (matrixB.height * matrixB.width) * sizeof(float), hipMemcpyHostToDevice);
        if (hipError_t != hipSuccess) {
            printf("hipMemcpy matrixB.h_rows -> matrixB.d_rows returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
            return 1;
        }

        hipError_t = hipMemcpy(matrixC.d_rows, matrixC.h_rows, (matrixC.height * matrixC.width) * sizeof(float), hipMemcpyHostToDevice);
        if (hipError_t != hipSuccess) {
            printf("hipMemcpy matrixC.h_rows -> matrixC.d_rows returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
            return 1;
        }
    }

    /* Scalar product of matrix A */
    printf("Executing scalar_matrix_mult(%5.1f, matrixA)...\n",scalar_value);
    gettimeofday(&start, NULL);
    if (!scalar_matrix_mult(scalar_value, &matrixA)) {
	    printf("%s: scalar_matrix_mult problem.", argv[0]);
	    return 1;
    }
    gettimeofday(&stop, NULL);
    printf("%f ms\n", timedifference_msec(start, stop));

    hipDeviceSynchronize();

    hipError_t = hipMemcpy(matrixA.h_rows, matrixA.d_rows, (matrixA.height * matrixA.width) * sizeof(float), hipMemcpyDeviceToHost);
    if (hipError_t != hipSuccess){
	    printf("hipMemcpy (d_y -> h_y) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
	    return 1;
    }

    /* Print matrix */
    printf("---------- Matrix A ----------\n");
    print_matrix(&matrixA);

    /* Write first result */
    printf("Writing first result: %s...\n", result1_filename);
    if (!store_matrix(&matrixA, result1_filename)) {
        printf("%s: failed to write first result to file.", argv[0]);
        return 1;
    }

    /* Calculate the product between matrix A and matrix B */
    printf("Executing matrix_matrix_mult(matrixA, matrixB, matrixC)...\n");
    gettimeofday(&start, NULL);
    if (!matrix_matrix_mult(&matrixA, &matrixB, &matrixC)) {
	    printf("%s: matrix_matrix_mult problem.", argv[0]);
	    return 1;
    }
    gettimeofday(&stop, NULL);
    printf("%f ms\n", timedifference_msec(start, stop));

    hipDeviceSynchronize();
    hipError_t = hipMemcpy(matrixC.h_rows, matrixC.d_rows, (matrixC.height * matrixC.width) * sizeof(float), hipMemcpyDeviceToHost);
    if (hipError_t != hipSuccess){
        printf("hipMemcpy (d_y -> h_y) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
        return 1;
    }

    /* Print matrix */
    printf("---------- Matrix C ----------\n");
    print_matrix(&matrixC);

    /* Write second result */
    printf("Writing second result: %s...\n", result2_filename);
    if (!store_matrix(&matrixC, result2_filename)) {
        printf("%s: failed to write second result to file.", argv[0]);
        return 1;
    }

    /* Check foor errors */
    printf("Checking matrixC for errors...\n");
    gettimeofday(&start, NULL);

    /*Para checar com a matriz 1024 X 1024 basta mudar o float da check_errors para 51200.00f que é o valor esperado para multiplicar as matrizes com 10.0 e 5.0 */	
    if (check_errors(&matrixC, 51200.0f) == 1){
        printf("No errors found\n");
    };

    gettimeofday(&stop, NULL);
    printf("%f ms\n", timedifference_msec(start, stop));

    hipFree(matrixA.d_rows);
    hipFree(matrixB.d_rows);
    hipFree(matrixC.d_rows);
    free(matrixA.h_rows);
    free(matrixB.h_rows);
    free(matrixC.h_rows);

    // Mark overall stop time
    gettimeofday(&overall_t2, NULL);

    // Show elapsed overall time
    printf("Overall time: %f ms\n", timedifference_msec(overall_t1, overall_t2));

    return 0;
}
