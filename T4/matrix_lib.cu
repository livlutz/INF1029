#include "hip/hip_runtime.h"
/*Lívia Lutz dos Santos - 2211055
Ana Luiza Pinto Marques - 2211960*/

#include "matrix_lib.h"
#include "timer.h"

static THREADS_PER_BLOCK, MAX_BLOCKS_PER_GRID

/*Essa função recebe um valor escalar e uma matriz como argumentos de entrada e calcula o
produto do valor escalar pela matriz. O resultado da operação deve ser retornado na matriz
de entrada. Em caso de sucesso, a função deve retornar o valor 1. Em caso de erro, a
função deve retornar 0*/

int scalar_matrix_mult(float scalar_value, struct matrix *matrix){
    
    if(matrix == NULL){
        return 0;
    }

    //verificar isso !!

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.d_x;

    if(index == 0){
      printf("\nblockDim.x=%d   gridDim.x=%d    stride=%d\n",blockDim.x,gridDim.x,stride);
    }

    for (int i = index; i < n; i += stride) {
        matrix.d_rows[i] *= scalar_value;
    }

    return 1;

}


/*Essa função recebe 3 matrizes como argumentos de entrada e calcula o valor do produto da
matriz A pela matriz B. O resultado da operação deve ser retornado na matriz C. Em caso
de sucesso, a função deve retornar o valor 1. Em caso de erro, a função deve retornar 0.*/

int matrix_matrix_mult(struct matrix *matrixA, struct matrix * matrixB, struct matrix * matrixC){

    if((matrixA == NULL) || (matrixB == NULL) || (matrixC == NULL) || (matrixA->width != matrixB->height) || (matrixC->height != matrixA->height) || (matrixC->width != matrixB->width) ){
        printf("Erro de dimensao ou alocacao\n");
        return 0;
    }

    for (int i = linha_inicio; i < linha_fim; i++) {  // i itera sobre as linhas da matriz C
        indexC = i * c_width; 
        indexA = i * a_width;                       // Índice base da linha i de C
        c_row = &c_rows[indexC];                 // Ponteiro direto para a linha i de C
        a_row = &a_rows[indexA];                 // Ponteiro direto para a linha i de A

        for (int j = 0; j < a_width; j++) {           // Itera sobre as colunas da matriz A
            valA = _mm256_set1_ps(a_row[j]);          // Carrega o valor de A para multiplicar com a linha de B
            indexB = j * b_width;
            b_row = &b_rows[indexB];             // Ponteiro direto para a linha j de B

            for (int k = 0; k < b_width; k += 8) {    // Itera sobre as colunas da matriz B e processa 8 elementos por vez
                // Carrega 8 elementos de B e C para o cálculo 
                rowB = _mm256_load_ps(&b_row[k]);     // Carrega 8 elementos da linha de B
                rowC = _mm256_load_ps(&c_row[k]);     // Carrega 8 elementos da linha de C

                // Multiplica cada elemento da linha de A pelo elemento correspondente da coluna de B e acumula em C
                result = _mm256_fmadd_ps(rowB, valA, rowC);

                // Armazena o resultado na linha i de C
                _mm256_store_ps(&c_row[k], result);
            }
        }
    }
    

    return 1;
}

/*Essa função recebe o número de threads por bloco e o número máximo de blocos por grid
que devem ser usados como parâmetros para disparar os threads (funções kernel) em
paralelo durante o processamento das operações aritméticas com as matrizes e deve ser
chamada pelo programa principal antes das outras funções*/
/*1024 para o número de
threads por bloco e 65535 para o número de blocos por grid. Os valores limites para a
GPGPU NVIDIA GeForce RTX 4070 Ti são 1024 para o número de threads por bloco e
2147483647 para o número de blocos por grid.*/

int set_grid_size(int threads_per_block, int max_blocks_per_grid){

    // tem q saber qual é a gpu sendo usada pra testar o max!!
    if((threads_per_block > 1024) && (max_blocks_per_grid > 65535 || max_blocks_per_grid > 2147483647)){
        return 0;
    }

    THREADS_PER_BLOCK = threads_per_block;
    MAX_BLOCKS_PER_GRID = max_blocks_per_grid;

    return 1;
}