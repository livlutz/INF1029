#include "hip/hip_runtime.h"
/*Lívia Lutz dos Santos - 2211055
Ana Luiza Pinto Marques - 2211960*/

#include "matrix_lib.h"
#include "timer.h"

static int THREADS_PER_BLOCK, MAX_BLOCKS_PER_GRID;

/*Essa função recebe um valor escalar e uma matriz como argumentos de entrada e calcula o
produto do valor escalar pela matriz utilizando CUDA. Cada função kernel deve calcular o
resultado do produto entre o valor escalar e um dos elementos da matriz (ou mais de um
elemento se o dataset for maior que o número de threads do GRID). O resultado da
operação deve ser retornado na matriz de entrada. Em caso de sucesso, a função deve
retornar o valor 1. Em caso de erro, a função deve retornar 0.*/

/*Cada função kernel deve calcular o
resultado do produto entre o valor escalar e um dos elementos da matriz (ou mais de um
elemento se o dataset for maior que o número de threads do GRID)*/
__global__
void scalar_mult(float scalar_value, float *d_rows, int matrix_size) {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < matrix_size; i += stride) {
        d_rows[i] *= scalar_value;
    }
}


int scalar_matrix_mult(float scalar_value, struct matrix *matrix) {
    
    if(matrix == NULL) {
        return 0;
    }

    int matrix_size = matrix->height * matrix->width;
    int threads_per_block = THREADS_PER_BLOCK;
    int blocks_per_grid =(matrix_size + threads_per_block - 1) / threads_per_block;

    scalar_mult<<<blocks_per_grid, threads_per_block>>>(scalar_value, matrix->d_rows, matrix_size);
    
    return 1;
}


/*Essa função recebe 3 matrizes como argumentos de entrada e calcula o valor do produto da
matriz A pela matriz B utilizando CUDA. Cada função kernel deve calcular o resultado
referente a um dos elementos da matriz C (ou mais de um elemento se o dataset for maior
que o número de threads do GRID). O resultado da operação deve ser retornado na matriz
C. Em caso de sucesso, a função deve retornar o valor 1. Em caso de erro, a função deve
retornar 0*/

/*Cada função kernel deve calcular o resultado
referente a um dos elementos da matriz C (ou mais de um elemento se o dataset for maior
que o número de threads do GRID).*/

__global__
void matrix_multiply(float *d_rowsA, float *d_rowsB, float *d_rowsC, unsigned long int C_height, unsigned long int A_width, unsigned long int B_width, unsigned long int C_width) {

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int column = blockIdx.y * blockDim.y + threadIdx.y;

    int indexC,linhaA;
    float result;

    if (row < C_height && column < B_width) {
        result = 0.0f;
        
        linhaA = row * A_width;
        indexC = row * C_width + column;
        // A multiplicação de matrizes é feita na soma dos produtos A[i, k] * B[k, j]
        // Para a célula C[row, column], percorre toda a dimensão k (do tamanho de A_width)
        for (int k = 0; k < A_width; k++) {
            result += d_rowsA[linhaA + k] * d_rowsB[k * B_width + column];
        }
        
        // Armazenar o resultado final na matriz C
        d_rowsC[indexC] = result;
    }
}

int matrix_matrix_mult(struct matrix *matrixA, struct matrix * matrixB, struct matrix * matrixC){

    if((matrixA == NULL) || (matrixB == NULL) || (matrixC == NULL) || (matrixA->width != matrixB->height) || (matrixC->height != matrixA->height) || (matrixC->width != matrixB->width) ){
        printf("Erro de dimensao ou alocacao\n");
        return 0;
    }

    float *d_rowsA = matrixA->d_rows;
    float *d_rowsB = matrixB->d_rows;
    float *d_rowsC = matrixC->d_rows;
    unsigned long int C_height = matrixC->height;
    unsigned long int A_width = matrixA->width;
    unsigned long int B_width = matrixB->width;
    unsigned long int C_width = matrixC->width;

    // Configuração do número de threads por bloco
    dim3 blockSize(THREADS_PER_BLOCK, THREADS_PER_BLOCK);  // 16x16 threads por bloco

    // Calcular o número de blocos necessários em ambas as dimensões (x e y)
    dim3 numBlocks((C_width + blockSize.x - 1) / blockSize.x, (C_height + blockSize.y - 1) / blockSize.y);

    if(numBlocks.x > MAX_BLOCKS_PER_GRID || numBlocks.y > MAX_BLOCKS_PER_GRID){
        numBlocks.x = MAX_BLOCKS_PER_GRID;
        numBlocks.y = MAX_BLOCKS_PER_GRID;
        return 0;
    }

    // Chamar o kernel de multiplicação de matrizes
    matrix_multiply<<<numBlocks, blockSize>>>(d_rowsA, d_rowsB, d_rowsC, C_height, A_width, B_width, C_width);

    return 1;
}

/*Essa função recebe o número de threads por bloco e o número máximo de blocos por grid
que devem ser usados como parâmetros para disparar os threads (funções kernel) em
paralelo durante o processamento das operações aritméticas com as matrizes e deve ser
chamada pelo programa principal antes das outras funções*/
/*1024 para o número de
threads por bloco e 65535 para o número de blocos por grid. Os valores limites para a
GPGPU NVIDIA GeForce RTX 4070 Ti são 1024 para o número de threads por bloco e
2147483647 para o número de blocos por grid.*/

int set_grid_size(int threads_per_block, int max_blocks_per_grid){

    if((threads_per_block > 1024) && (max_blocks_per_grid > 2147483647)){
        return 0;
    }

    THREADS_PER_BLOCK = threads_per_block;
    MAX_BLOCKS_PER_GRID = max_blocks_per_grid;

    return 1;
}