#include "hip/hip_runtime.h"
/*Lívia Lutz dos Santos - 2211055
Ana Luiza Pinto Marques - 2211960*/

#include "matrix_lib.h"
#include "timer.h"

static int THREADS_PER_BLOCK, MAX_BLOCKS_PER_GRID;

/*Essa função recebe um valor escalar e uma matriz como argumentos de entrada e calcula o
produto do valor escalar pela matriz utilizando CUDA. Cada função kernel deve calcular o
resultado do produto entre o valor escalar e um dos elementos da matriz (ou mais de um
elemento se o dataset for maior que o número de threads do GRID). O resultado da
operação deve ser retornado na matriz de entrada. Em caso de sucesso, a função deve
retornar o valor 1. Em caso de erro, a função deve retornar 0.*/
__global__
void scalar_mult(float scalar_value, struct matrix *matrix){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if(index == 0){
      printf("\nblockDim.x=%d   gridDim.x=%d    stride=%d\n",blockDim.x,gridDim.x,stride);

    }

    for(int i = index;i < (matrix->height * matrix->width); i += stride){
        matrix->d_rows[i] *= scalar_value;
    }
}

int scalar_matrix_mult(float scalar_value, struct matrix *matrix){
    
    if(matrix == NULL){
        return 0;
    }

    scalar_mult<<<MAX_BLOCKS_PER_GRID,THREADS_PER_BLOCK>>>(scalar_value,matrix);

    return 1;
}

/*Essa função recebe 3 matrizes como argumentos de entrada e calcula o valor do produto da
matriz A pela matriz B utilizando CUDA. Cada função kernel deve calcular o resultado
referente a um dos elementos da matriz C (ou mais de um elemento se o dataset for maior
que o número de threads do GRID). O resultado da operação deve ser retornado na matriz
C. Em caso de sucesso, a função deve retornar o valor 1. Em caso de erro, a função deve
retornar 0*/
__global__
void matrix_multiply(struct matrix *matrixA, struct matrix * matrixB, struct matrix * matrixC){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int indexA, indexB, indexC;
    float valA;
    float*rowB,*rowC,result;

    if(index == 0){
      printf("\nblockDim.x=%d   gridDim.x=%d    stride=%d\n",blockDim.x,gridDim.x,stride);
    }

    for(int i = index; i < matrixC->height; i += stride){

        for(int j = 0; j < matrixA->width;j++){

            indexA = i * matrixA->width + j;

            valA = matrixA->d_rows[indexA];

            for (int k = 0; k < matrixB->width; k ++){
                //Calcula posicao inicial do indice da matrizB
                indexB = j * matrixB->width + k;

                //Calcula posicao inicial dos indices da matrizC aqui e depois incrementa o valor dentro do loop
                indexC = i * matrixC->width + k;

                //Calcula o valor da linha da matrizB
                rowB = &matrixB->d_rows[indexB];

                //Calcula o valor da linha da matrizC
                rowC = &matrixC->d_rows[indexC];

                rowC[k] += valA * rowB[k];

            }
        }
        
    }
}

int matrix_matrix_mult(struct matrix *matrixA, struct matrix * matrixB, struct matrix * matrixC){

    if((matrixA == NULL) || (matrixB == NULL) || (matrixC == NULL) || (matrixA->width != matrixB->height) || (matrixC->height != matrixA->height) || (matrixC->width != matrixB->width) ){
        printf("Erro de dimensao ou alocacao\n");
        return 0;
    }

    matrix_multiply<<<MAX_BLOCKS_PER_GRID,THREADS_PER_BLOCK>>>(matrixA,matrixB,matrixC);
   
    return 1;
}

/*Essa função recebe o número de threads por bloco e o número máximo de blocos por grid
que devem ser usados como parâmetros para disparar os threads (funções kernel) em
paralelo durante o processamento das operações aritméticas com as matrizes e deve ser
chamada pelo programa principal antes das outras funções*/
/*1024 para o número de
threads por bloco e 65535 para o número de blocos por grid. Os valores limites para a
GPGPU NVIDIA GeForce RTX 4070 Ti são 1024 para o número de threads por bloco e
2147483647 para o número de blocos por grid.*/

int set_grid_size(int threads_per_block, int max_blocks_per_grid){

    // tem q saber qual é a gpu sendo usada pra testar o max!!
    if((threads_per_block > 1024) && (max_blocks_per_grid > 2147483647)){
        return 0;
    }

    THREADS_PER_BLOCK = threads_per_block;
    MAX_BLOCKS_PER_GRID = max_blocks_per_grid;

    return 1;
}