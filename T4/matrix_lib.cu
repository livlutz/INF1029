#include "hip/hip_runtime.h"
/*Lívia Lutz dos Santos - 2211055
Ana Luiza Pinto Marques - 2211960*/

#include "matrix_lib.h"
#include "timer.h"

static int THREADS_PER_BLOCK, MAX_BLOCKS_PER_GRID;

/*Essa função recebe um valor escalar e uma matriz como argumentos de entrada e calcula o
produto do valor escalar pela matriz utilizando CUDA. Cada função kernel deve calcular o
resultado do produto entre o valor escalar e um dos elementos da matriz (ou mais de um
elemento se o dataset for maior que o número de threads do GRID). O resultado da
operação deve ser retornado na matriz de entrada. Em caso de sucesso, a função deve
retornar o valor 1. Em caso de erro, a função deve retornar 0.*/

/*Cada função kernel deve calcular o
resultado do produto entre o valor escalar e um dos elementos da matriz (ou mais de um
elemento se o dataset for maior que o número de threads do GRID)*/
__global__
void scalar_mult(float scalar_value, float *d_rows, int matrix_size) {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < matrix_size; i += stride) {
        d_rows[i] *= scalar_value;
    }
}


int scalar_matrix_mult(float scalar_value, struct matrix *matrix) {
    
    if(matrix == NULL) {
        return 0;
    }

    int matrix_size = matrix->height * matrix->width;
    int threads_per_block = THREADS_PER_BLOCK;
    int blocks_per_grid =(matrix_size + threads_per_block - 1) / threads_per_block;

    scalar_mult<<<blocks_per_grid, threads_per_block>>>(scalar_value, matrix->d_rows, matrix_size);
    
    return 1;
}


/*Essa função recebe 3 matrizes como argumentos de entrada e calcula o valor do produto da
matriz A pela matriz B utilizando CUDA. Cada função kernel deve calcular o resultado
referente a um dos elementos da matriz C (ou mais de um elemento se o dataset for maior
que o número de threads do GRID). O resultado da operação deve ser retornado na matriz
C. Em caso de sucesso, a função deve retornar o valor 1. Em caso de erro, a função deve
retornar 0*/

/*Cada função kernel deve calcular o resultado
referente a um dos elementos da matriz C (ou mais de um elemento se o dataset for maior
que o número de threads do GRID).*/

__global__
void matrix_multiply(float *d_rowsA, float *d_rowsB, float *d_rowsC, unsigned long int C_height, unsigned long int A_width, unsigned long int B_width, unsigned long int C_width) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int indexA, indexB, indexC;
    float valA,rowB,rowC,result;

    //itera por linhas da matriz C
    for (int i = index; i < C_height; i += stride) {

        //itera por colunas da matriz A
        for (int j = 0; j < A_width; j++) {

            //Calcula posicao inicial do indice da matrizA 
            indexA = i * A_width + j;

            //valor do elemento da matriz A
            valA = d_rowsA[indexA];

            //itera por linhas da matriz B
            for (int k = 0; k < B_width; k++) {

                //Calcula posicao inicial do indice da matrizB
                indexB = j * B_width + k;
                //Calcula posicao inicial dos indices da matrizC aqui e depois incrementa o valor dentro do loop
                indexC = i * C_width + k;

                rowB = d_rowsB[indexB];

                rowC = d_rowsC[indexC];

                //Calcula o valor do elemento da matriz C
                result = rowC + valA * rowB;

                //Atualiza o valor do elemento da matriz C
                d_rowsC[indexC] = result;

            }

        }
    }
}

int matrix_matrix_mult(struct matrix *matrixA, struct matrix * matrixB, struct matrix * matrixC){

    if((matrixA == NULL) || (matrixB == NULL) || (matrixC == NULL) || (matrixA->width != matrixB->height) || (matrixC->height != matrixA->height) || (matrixC->width != matrixB->width) ){
        printf("Erro de dimensao ou alocacao\n");
        return 0;
    }

    float *d_rowsA = matrixA->d_rows;
    float *d_rowsB = matrixB->d_rows;
    float *d_rowsC = matrixC->d_rows;
    unsigned long int C_height = matrixC->height;
    unsigned long int A_width = matrixA->width;
    unsigned long int B_width = matrixB->width;
    unsigned long int C_width = matrixC->width;

    int blockSize = THREADS_PER_BLOCK;
    int numBlocks = ((matrixC->height * matrixC->width) + blockSize - 1) / blockSize;
    if (numBlocks > MAX_BLOCKS_PER_GRID) numBlocks = MAX_BLOCKS_PER_GRID;
    
    matrix_multiply<<<numBlocks,blockSize>>>(d_rowsA,d_rowsB,d_rowsC,C_height,A_width,B_width,C_width);
    return 1;
}

/*Essa função recebe o número de threads por bloco e o número máximo de blocos por grid
que devem ser usados como parâmetros para disparar os threads (funções kernel) em
paralelo durante o processamento das operações aritméticas com as matrizes e deve ser
chamada pelo programa principal antes das outras funções*/
/*1024 para o número de
threads por bloco e 65535 para o número de blocos por grid. Os valores limites para a
GPGPU NVIDIA GeForce RTX 4070 Ti são 1024 para o número de threads por bloco e
2147483647 para o número de blocos por grid.*/

int set_grid_size(int threads_per_block, int max_blocks_per_grid){

    if((threads_per_block > 1024) && (max_blocks_per_grid > 2147483647)){
        return 0;
    }

    THREADS_PER_BLOCK = threads_per_block;
    MAX_BLOCKS_PER_GRID = max_blocks_per_grid;

    return 1;
}