#include "hip/hip_runtime.h"
/*Lívia Lutz dos Santos - 2211055
Ana Luiza Pinto Marques - 2211960*/

#include "matrix_lib.h"
#include "timer.h"

static int THREADS_PER_BLOCK, MAX_BLOCKS_PER_GRID;

/*Essa função recebe um valor escalar e uma matriz como argumentos de entrada e calcula o
produto do valor escalar pela matriz utilizando CUDA. Cada função kernel deve calcular o
resultado do produto entre o valor escalar e um dos elementos da matriz (ou mais de um
elemento se o dataset for maior que o número de threads do GRID). O resultado da
operação deve ser retornado na matriz de entrada. Em caso de sucesso, a função deve
retornar o valor 1. Em caso de erro, a função deve retornar 0.*/
__global__
void scalar_mult(float scalar_value, float *d_rows, int matrix_size) {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if(index == 0){
        printf("\nblockDim.x=%d   gridDim.x=%d    stride=%d\n",blockDim.x,gridDim.x,stride);
    }

    for(int i = index; i < matrix_size; i += stride) {
        d_rows[i] *= scalar_value;
    }
}


int scalar_matrix_mult(float scalar_value, struct matrix *matrix) {
    if(matrix == NULL) {
        return 0;
    }

    int matrix_size = matrix->height * matrix->width;
    scalar_mult<<<MAX_BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(scalar_value, matrix->d_rows, matrix_size);

    return 1;
}


/*Essa função recebe 3 matrizes como argumentos de entrada e calcula o valor do produto da
matriz A pela matriz B utilizando CUDA. Cada função kernel deve calcular o resultado
referente a um dos elementos da matriz C (ou mais de um elemento se o dataset for maior
que o número de threads do GRID). O resultado da operação deve ser retornado na matriz
C. Em caso de sucesso, a função deve retornar o valor 1. Em caso de erro, a função deve
retornar 0*/

__global__
void matrix_multiply(float *d_rowsA, float *d_rowsB, float *d_rowsC, unsigned long int C_height, unsigned long int A_width, unsigned long int B_width, unsigned long int C_width) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    float sum;
    int indexA, indexB, indexC;

    // Calcular a multiplicação por bloco e por thread, onde cada thread cuida de um elemento de matrixC
    for (int i = index; i < C_height; i += stride) {
        for (int k = 0; k < A_width; k++) {
            sum = 0.0f;

            for (int j = 0; j < B_width; j++) {
                indexA = i * A_width + j;
                indexB = j * B_width + k;

                // Acumula o produto de A e B
                sum += d_rowsA[indexA] * d_rowsB[indexB];
            }

            // Armazena o valor final na posição (i, k) de C
            indexC = i * C_width+ k;
            d_rowsC[indexC] = sum;
        }
    }
}

int matrix_matrix_mult(struct matrix *matrixA, struct matrix * matrixB, struct matrix * matrixC){

    if((matrixA == NULL) || (matrixB == NULL) || (matrixC == NULL) || (matrixA->width != matrixB->height) || (matrixC->height != matrixA->height) || (matrixC->width != matrixB->width) ){
        printf("Erro de dimensao ou alocacao\n");
        return 0;
    }

    float *d_rowsA = matrixA->d_rows;
    float *d_rowsB = matrixB->d_rows;
    float *d_rowsC = matrixC->d_rows;
    unsigned long int C_height = matrixC->height;
    unsigned long int A_width = matrixA->width;
    unsigned long int B_width = matrixB->width;
    unsigned long int C_width = matrixC->width;

    matrix_multiply<<<MAX_BLOCKS_PER_GRID,THREADS_PER_BLOCK>>>(d_rowsA,d_rowsB,d_rowsC,C_height,A_width,B_width,C_width);
   
    return 1;
}

/*Essa função recebe o número de threads por bloco e o número máximo de blocos por grid
que devem ser usados como parâmetros para disparar os threads (funções kernel) em
paralelo durante o processamento das operações aritméticas com as matrizes e deve ser
chamada pelo programa principal antes das outras funções*/
/*1024 para o número de
threads por bloco e 65535 para o número de blocos por grid. Os valores limites para a
GPGPU NVIDIA GeForce RTX 4070 Ti são 1024 para o número de threads por bloco e
2147483647 para o número de blocos por grid.*/

int set_grid_size(int threads_per_block, int max_blocks_per_grid){

    if((threads_per_block > 1024) && (max_blocks_per_grid > 2147483647)){
        return 0;
    }

    THREADS_PER_BLOCK = threads_per_block;
    MAX_BLOCKS_PER_GRID = max_blocks_per_grid;

    return 1;
}