#include "hip/hip_runtime.h"
/*Lívia Lutz dos Santos - 2211055
Ana Luiza Pinto Marques - 2211960*/

#include "matrix_lib.h"
#include "timer.h"

static int THREADS_PER_BLOCK, MAX_BLOCKS_PER_GRID;

/*Essa função recebe um valor escalar e uma matriz como argumentos de entrada e calcula o
produto do valor escalar pela matriz utilizando CUDA. Cada função kernel deve calcular o
resultado do produto entre o valor escalar e um dos elementos da matriz (ou mais de um
elemento se o dataset for maior que o número de threads do GRID). O resultado da
operação deve ser retornado na matriz de entrada. Em caso de sucesso, a função deve
retornar o valor 1. Em caso de erro, a função deve retornar 0.*/
__global__
void scalar_mult(float scalar_value, float *d_rows, int matrix_size) {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < matrix_size; i += stride) {
        d_rows[i] *= scalar_value;
    }
}


int scalar_matrix_mult(float scalar_value, struct matrix *matrix) {
    if(matrix == NULL) {
        return 0;
    }

    hipError_t hipError_t;
    int matrix_size = matrix->height * matrix->width;
    int threads_per_block = THREADS_PER_BLOCK;
    int blocks_per_grid =(matrix_size + threads_per_block - 1) / threads_per_block;

    if(matrix->alloc_mode == 0){
        for(int i = 0; i < matrix->height; i++){
            hipError_t = hipMemcpy(matrix->d_rows, &matrix->h_rows[i * matrix->width], matrix->width * sizeof(float), hipMemcpyHostToDevice);
            if (hipError_t != hipSuccess) {
                printf("hipMemcpy h_rowA -> d_rowA returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
                hipFree(matrix->d_rows);
                return 0;
            }

            scalar_mult<<<blocks_per_grid, threads_per_block>>>(scalar_value, matrix->d_rows, matrix_size);
            hipDeviceSynchronize();

            hipError_t = hipMemcpy(&matrix->h_rows[i * matrix->width], matrix->d_rows, matrix->width * sizeof(float), hipMemcpyDeviceToHost);
            if (hipError_t != hipSuccess) {
                printf("hipMemcpy d_rowA -> h_rowA returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
                hipFree(matrix->d_rows);
                return 0;
            }
        }

        return 1;
    }

    else{
        scalar_mult<<<blocks_per_grid, threads_per_block>>>(scalar_value, matrix->d_rows, matrix_size);
        hipDeviceSynchronize();
        return 1;
    }
   
    return 0;
}


/*Essa função recebe 3 matrizes como argumentos de entrada e calcula o valor do produto da
matriz A pela matriz B utilizando CUDA. Cada função kernel deve calcular o resultado
referente a um dos elementos da matriz C (ou mais de um elemento se o dataset for maior
que o número de threads do GRID). O resultado da operação deve ser retornado na matriz
C. Em caso de sucesso, a função deve retornar o valor 1. Em caso de erro, a função deve
retornar 0*/

__global__
void matrix_multiply(float *d_rowsA, float *d_rowsB, float *d_rowsC, unsigned long int C_height, unsigned long int A_width, unsigned long int B_width, unsigned long int C_width) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int indexA, indexB, indexC;
    float valA,rowB,rowC,result;

    //itera por linhas da matriz C
    for (int i = index; i < C_height; i += stride) {

        //itera por colunas da matriz A
        for (int j = 0; j < A_width; j++) {

            //Calcula posicao inicial do indice da matrizA 
            indexA = i * A_width + j;

            //valor do elemento da matriz A
            valA = d_rowsA[indexA];

            //itera por linhas da matriz B
            for (int k = 0; k < B_width; k++) {

                //Calcula posicao inicial do indice da matrizB
                indexB = j * B_width + k;
                //Calcula posicao inicial dos indices da matrizC aqui e depois incrementa o valor dentro do loop
                indexC = i * C_width + k;

                rowB = d_rowsB[indexB];

                rowC = d_rowsC[indexC];

                //Calcula o valor do elemento da matriz C
                result = rowC + valA * rowB;

                //Atualiza o valor do elemento da matriz C
                d_rowsC[indexC] = result;

            }

        }
    }
}

int matrix_matrix_mult(struct matrix *matrixA, struct matrix * matrixB, struct matrix * matrixC){

    if((matrixA == NULL) || (matrixB == NULL) || (matrixC == NULL) || (matrixA->width != matrixB->height) || (matrixC->height != matrixA->height) || (matrixC->width != matrixB->width) ){
        printf("Erro de dimensao ou alocacao\n");
        return 0;
    }

    hipError_t hipError_t;
    float *d_rowsA = matrixA->d_rows;
    float *d_rowsB = matrixB->d_rows;
    float *d_rowsC = matrixC->d_rows;
    unsigned long int C_height = matrixC->height;
    unsigned long int A_width = matrixA->width;
    unsigned long int B_width = matrixB->width;
    unsigned long int C_width = matrixC->width;

    int blockSize = THREADS_PER_BLOCK;
    int numBlocks = ((matrixC->height * matrixC->width) + blockSize - 1) / blockSize;
    if (numBlocks > MAX_BLOCKS_PER_GRID) numBlocks = MAX_BLOCKS_PER_GRID;

    if(matrixA->alloc_mode == 0 && matrixC->alloc_mode == 0){
        // Processa cada linha de A e C individualmente
        for (int i = 0; i < matrixA->height; i++) {
            // Copia a linha atual de A para o device
            hipError_t = hipMemcpy(d_rowsA, &matrixA->h_rows[i * matrixA->width], matrixA->width * sizeof(float), hipMemcpyHostToDevice);
            if (hipError_t != hipSuccess) {
                printf("hipMemcpy h_rowA -> d_rowA returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
                hipFree(d_rowsA);
                hipFree(d_rowsC);
                return 0;
            }

            // Copia a linha atual de C para o device
            hipError_t = hipMemcpy(d_rowsC, &matrixC->h_rows[i * matrixC->width], matrixC->width * sizeof(float), hipMemcpyHostToDevice);
            if (hipError_t != hipSuccess) {
                printf("hipMemcpy h_rowC -> d_rowC returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
                hipFree(d_rowsA);
                hipFree(d_rowsC);
                return 0;
            }

            // Executa o kernel para calcular a linha `i` de C
            matrix_multiply<<<numBlocks, blockSize>>>(d_rowsA, matrixB->d_rows, d_rowsC, 1, matrixA->width, matrixB->width, matrixC->width);
            hipDeviceSynchronize();

            // Copia o resultado da linha `i` de C do device de volta para a host
            hipError_t = hipMemcpy(&matrixC->h_rows[i * matrixC->width], d_rowsC, matrixC->width * sizeof(float), hipMemcpyDeviceToHost);
            if (hipError_t != hipSuccess) {
                printf("hipMemcpy d_rowC -> h_rowC returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
                hipFree(d_rowsA);
                hipFree(d_rowsC);
                return 0;
            }

            // Copia a linha atual de A de volta para a host
            hipError_t = hipMemcpy(&matrixA->h_rows[i * matrixA->width], d_rowsA, matrixA->width * sizeof(float), hipMemcpyDeviceToHost);
            if (hipError_t != hipSuccess) {
                printf("hipMemcpy d_rowA -> h_rowA returned error %s (code %d)\n", hipGetErrorString(hipError_t), hipError_t);
                hipFree(d_rowsA);
                hipFree(d_rowsC);
                return 0;
            }
        }
    }

    else{
        matrix_multiply<<<numBlocks,blockSize>>>(d_rowsA,d_rowsB,d_rowsC,C_height,A_width,B_width,C_width);
        hipDeviceSynchronize();
        return 1;
    }

    return 1;
}

/*Essa função recebe o número de threads por bloco e o número máximo de blocos por grid
que devem ser usados como parâmetros para disparar os threads (funções kernel) em
paralelo durante o processamento das operações aritméticas com as matrizes e deve ser
chamada pelo programa principal antes das outras funções*/
/*1024 para o número de
threads por bloco e 65535 para o número de blocos por grid. Os valores limites para a
GPGPU NVIDIA GeForce RTX 4070 Ti são 1024 para o número de threads por bloco e
2147483647 para o número de blocos por grid.*/

int set_grid_size(int threads_per_block, int max_blocks_per_grid){

    if((threads_per_block > 1024) && (max_blocks_per_grid > 2147483647)){
        return 0;
    }

    THREADS_PER_BLOCK = threads_per_block;
    MAX_BLOCKS_PER_GRID = max_blocks_per_grid;

    return 1;
}