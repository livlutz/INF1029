#include "hip/hip_runtime.h"
/*Lívia Lutz dos Santos - 2211055
Ana Luiza Pinto Marques - 2211960*/

#include "matrix_lib.h"
#include "timer.h"

static int THREADS_PER_BLOCK, MAX_BLOCKS_PER_GRID;

/*Essa função recebe um valor escalar e uma matriz como argumentos de entrada e calcula o
produto do valor escalar pela matriz utilizando CUDA. Cada função kernel deve calcular o
resultado do produto entre o valor escalar e um dos elementos da matriz (ou mais de um
elemento se o dataset for maior que o número de threads do GRID). O resultado da
operação deve ser retornado na matriz de entrada. Em caso de sucesso, a função deve
retornar o valor 1. Em caso de erro, a função deve retornar 0.*/

/*Cada função kernel deve calcular o
resultado do produto entre o valor escalar e um dos elementos da matriz (ou mais de um
elemento se o dataset for maior que o número de threads do GRID)*/
__global__
void scalar_mult(float scalar_value, float *d_rows, int matrix_size) {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < matrix_size; i += stride) {
        d_rows[i] *= scalar_value;
    }
}


int scalar_matrix_mult(float scalar_value, struct matrix *matrix) {
    
    if(matrix == NULL) {
        return 0;
    }

    int matrix_size = matrix->height * matrix->width;
    int threads_per_block = THREADS_PER_BLOCK;
    int blocks_per_grid =(matrix_size + threads_per_block - 1) / threads_per_block;

    scalar_mult<<<blocks_per_grid, threads_per_block>>>(scalar_value, matrix->d_rows, matrix_size);

    hipDeviceSynchronize();
    
    return 1;
}


/*Essa função recebe 3 matrizes como argumentos de entrada e calcula o valor do produto da
matriz A pela matriz B utilizando CUDA. Cada função kernel deve calcular o resultado
referente a um dos elementos da matriz C (ou mais de um elemento se o dataset for maior
que o número de threads do GRID). O resultado da operação deve ser retornado na matriz
C. Em caso de sucesso, a função deve retornar o valor 1. Em caso de erro, a função deve
retornar 0*/

/*Cada função kernel deve calcular o resultado
referente a um dos elementos da matriz C (ou mais de um elemento se o dataset for maior
que o número de threads do GRID).*/

__global__
void matrix_multiply(int matrixA_alloc_mode, int matrixB_alloc_mode, int matrixC_alloc_mode, unsigned long int matrixA_height, unsigned long int matrixA_width, unsigned long int matrixB_height, unsigned long int matrixB_width, unsigned long int matrixC_height, unsigned long int matrixC_width, float *matrixA_rows, float *matrixB_rows, float *matrixC_rows) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;    

    int indexC,linhaA;
    float result;

    if(matrixA_alloc_mode == 0 && matrixB_alloc_mode == 1 && matrixC_alloc_mode == 0){
        for (int i = index; i < matrixC_width; i += stride) {
            matrixC_rows[i] += matrixA_rows[i] * matrixB_rows[i];
        }
    }

    else if(matrixA_alloc_mode == 1 && matrixB_alloc_mode == 1 && matrixC_alloc_mode == 1){

        for(int i = index; i < matrixA_height; i += stride){

            linhaA = i * matrixA_width;

            indexC = i * matrixC_width;

            for(int j = 0; j < matrixB_width; j++){

                result = 0;

                for(int k = 0; k < matrixA_width; k++){

                    result += matrixA_rows[linhaA + k] * matrixB_rows[k * matrixB_width + j];
                }

                matrixC_rows[indexC + j] = result;
            }

        }
    }
}

int matrix_matrix_mult(struct matrix *matrixA, struct matrix * matrixB, struct matrix * matrixC){

    if((matrixA == NULL) || (matrixB == NULL) || (matrixC == NULL) || (matrixA->width != matrixB->height) || (matrixC->height != matrixA->height) || (matrixC->width != matrixB->width) ){
        printf("Erro de dimensao ou alocacao\n");
        return 0;
    }
    
    int numBlocks,blockSize,loop_limit,chunk_size;
    numBlocks = MAX_BLOCKS_PER_GRID;
    blockSize = THREADS_PER_BLOCK;

    int A_alloc_mode = matrixA->alloc_mode, B_alloc_mode = matrixB->alloc_mode, C_alloc_mode = matrixC->alloc_mode;
    unsigned long int A_height = matrixA->height, A_width = matrixA->width, B_height = matrixB->height, B_width = matrixB->width, C_height = matrixC->height, C_width = matrixC->width;
    float *A_rows = matrixA->d_rows, *B_rows = matrixB->d_rows, *C_rows = matrixC->d_rows;

    if(matrixA->alloc_mode == 0 && matrixC->alloc_mode == 0){
        loop_limit = ((matrixA.height * matrixA.width) + max_mem_gpu - 1)/max_mem_gpu;
        chunk_size = max_mem_gpu;

        for(int count = 0; count < loop_limit; count++){
            if((matrixA.height * matrixA.width) % max_mem_gpu != 0 && count == loop_limit -1){
                chunk_size = (matrixA.height * matrixA.width) % max_mem_gpu;
            }
        matrix_multiply<<<numBlocks, blockSize>>>(A_alloc_mode, B_alloc_mode, C_alloc_mode, A_height, A_width, B_height, B_width, C_height, C_width, A_rows, B_rows, C_rows);
        hipDeviceSynchronize();
    }
    // Chamar o kernel de multiplicação de matrizes
    matrix_multiply<<<numBlocks, blockSize>>>(A_alloc_mode, B_alloc_mode, C_alloc_mode, A_height, A_width, B_height, B_width, C_height, C_width, A_rows, B_rows, C_rows);

    hipDeviceSynchronize();

    return 1;
}

/*Essa função recebe o número de threads por bloco e o número máximo de blocos por grid
que devem ser usados como parâmetros para disparar os threads (funções kernel) em
paralelo durante o processamento das operações aritméticas com as matrizes e deve ser
chamada pelo programa principal antes das outras funções*/
/*1024 para o número de
threads por bloco e 65535 para o número de blocos por grid. Os valores limites para a
GPGPU NVIDIA GeForce RTX 4070 Ti são 1024 para o número de threads por bloco e
2147483647 para o número de blocos por grid.*/

int set_grid_size(int threads_per_block, int max_blocks_per_grid){

    if((threads_per_block > 1024) && (max_blocks_per_grid > 2147483647)){
        return 0;
    }

    THREADS_PER_BLOCK = threads_per_block;
    MAX_BLOCKS_PER_GRID = max_blocks_per_grid;

    return 1;
}